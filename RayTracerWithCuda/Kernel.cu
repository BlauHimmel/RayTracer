
#include "hip/hip_runtime.h"


#include <math.h>
#include <stdio.h>

__device__ unsigned short RD(int i, int j, int W, int H) {
#define D(x) (x-W/2.)/(W/2.)
	float x = D(i), y = D(j), X, Y, n = 0; while (n++<200 && (X = x*x) + (Y = y*y)<4) { x = X - Y + .36237; y = 2 * x*y + .32; }return log(n) * 256;
}

__device__ unsigned short GR(int i, int j, int W, int H) {
	float x = D(i), y = D(j), X, Y, n = 0; while (n++<200 && (x*x + y*y)<4) { X = x; Y = y; x = X*X - Y*Y + -.7; y = 2 * X*Y + .27015; }return log(n) * 128;
}

__device__ unsigned short BL(int i, int j, int W, int H) {
	float x = D(i), y = D(j), X, Y, n = 0; while (n++<600 && (x*x + y*y)<4) { X = x; Y = y; x = X*X - Y*Y + .36237; y = 2 * X*Y + .32; }return log(n) * 128;
}

__global__ void RGBKernel(float* R, float* G, float* B, int W, int H)
{
	int x = blockIdx.x;
	int y = threadIdx.x;

	R[x * W + y] = RD(x, y, W, H) / 255.0f;
	G[x * W + y] = GR(x, y, W, H) / 255.0f;
	B[x * W + y] = BL(x, y, W, H) / 255.0f;
}

extern "C" hipError_t cuadRGB(int W, int H, float* R, float* G, float* B)
{
	float* DevR;
	float* DevG;
	float* DevB;

	auto cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)(&DevR), sizeof(float) * W * H);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)(&DevG), sizeof(float) * W * H);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)(&DevB), sizeof(float) * W * H);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	RGBKernel<<<W, H>>> (DevR, DevG, DevB, W, H);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(R, DevR, sizeof(float) * W * H, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(G, DevG, sizeof(float) * W * H, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(B, DevB, sizeof(float) * W * H, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(DevR);
	hipFree(DevG);
	hipFree(DevB);

	return cudaStatus;
}